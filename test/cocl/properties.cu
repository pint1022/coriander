// test calling kernels from different threads, in parallel (can be different kernels, or same.  either way, should work, not crash :-) )

#include <iostream>
#include <memory>
#include <cassert>
#include <sstream>

using namespace std;

#include <hip/hip_runtime.h>

const int N = 1024;

int main(int argc, char *argv[]) {
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    cout << "maxworkgroupsize " << prop.maxThreadsPerBlock << endl;
    size_t free;
    size_t total;
    hipMemGetInfo(&free, &total);
    cout << "free " << free << " total " << total << endl;
    return 0;
}
