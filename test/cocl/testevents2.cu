// tests cuEventCreate

#include <iostream>
#include <memory>

using namespace std;

#include <hip/hip_runtime.h>

__global__ void longKernel(float *data, int N, float value) {
    for(int i = 0; i < N; i++) {
        data[i] += value;
    }
}

int main(int argc, char *argv[]) {
    int N = 202400; // * 1024;

    float *hostfloats = new float[N];
    float *gpufloats;
    hipMalloc((void **)&gpufloats, N * sizeof(float));

    longKernel<<<dim3(102400 / 32, 1, 1), dim3(32, 1, 1)>>>(gpufloats, N, 3.0f);
    cout << "queued kernel x" << endl;

    hipEvent_t event;
    hipEventCreateWithFlags(&event, hipEventDisableTiming);
    hipStream_t stream;
    hipStreamCreateWithFlags(&stream, 0);

    cout << "created event" << endl;

    for(int i = 0; i < 5; i++) {
        if(i % 10 == 0) {
            cout << "i " << i << endl;
        }
        longKernel<<<dim3(102400 / 32, 1, 1), dim3(32, 1, 1)>>>(gpufloats, N, 3.0f);
        // cout << "queued kernel x" << endl;

        hipEventRecord(event, stream);
        // cout << "recoreded event" << endl;

        // cout << "event finished? " << (cuEventQuery(event) == 0) << endl;

        hipEventSynchronize(event);
        // cout << "synchronized event" << endl;

        // cout << "event finished? " << (cuEventQuery(event) == 0) << endl;
    }

    hipStreamDestroy(stream);
    hipEventDestroy(event);
    hipFree(gpufloats);

    cout << "finished" << endl;

    return 0;
}
