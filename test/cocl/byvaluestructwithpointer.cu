#include <iostream>
#include <memory>
#include <cassert>

using namespace std;

#include <hip/hip_runtime.h>

struct MyStruct {
    float *floats;
    float afloat;
};

struct MyStruct2 {
    float *floats1;
    float *floats2;
    float afloat;
};

__global__ void getValue(struct MyStruct mystruct, float *data) {
    data[0] = mystruct.floats[0] + 3.0f;
}

__global__ void getValue2(struct MyStruct2 mystruct, float *data1, float *data2) {
    data1[0] = mystruct.floats1[0] + 3.0f;
    data2[0] = mystruct.floats2[0] + 5.0f;
}

void test1(hipStream_t &stream) {
    int N = 1024;

    // CUstream stream;
    // cuStreamCreate(&stream, 0);

    float *hostFloats1;
    float *hostFloats2;
    hipHostAlloc((void **)&hostFloats1, N * sizeof(float), hipHostMallocPortable);
    hipHostAlloc((void **)&hostFloats2, N * sizeof(float), hipHostMallocPortable);

    hipDeviceptr_t deviceFloats1;
    hipDeviceptr_t deviceFloats2;
    hipMalloc(&deviceFloats1, N * sizeof(float));
    hipMalloc(&deviceFloats2, N * sizeof(float));

    MyStruct mystruct;
    mystruct.floats = (float *)deviceFloats1;
    hostFloats1[0] = 123;

    hipMemcpyHtoDAsync(
        (hipDeviceptr_t)(((float *)deviceFloats1)),
        hostFloats1,
        N * sizeof(float),
        stream
    );

    getValue<<<dim3(1,1,1), dim3(32,1,1), 0, stream>>>(mystruct, ((float *)deviceFloats2) + 0);

    // now copy back entire buffer
    // hostFloats[64] = 0.0f;
    hipMemcpyDtoHAsync(hostFloats2, deviceFloats2, N * sizeof(float), stream);
    hipStreamSynchronize(stream);

    // and check the values...
    cout << hostFloats2[0] << endl;

    assert(hostFloats2[0] == 126);

    hipHostFree(hostFloats1);
    hipHostFree(hostFloats2);
    hipFree(deviceFloats1);
    hipFree(deviceFloats2);
    // cuStreamDestroy(stream);
}

void test2(hipStream_t &stream) {
    int N = 1024;

    float *hostFloats1;
    float *hostFloats2;
    float *hostFloats3;
    float *hostFloats4;
    hipHostAlloc((void **)&hostFloats1, N * sizeof(float), hipHostMallocPortable);
    hipHostAlloc((void **)&hostFloats2, N * sizeof(float), hipHostMallocPortable);
    hipHostAlloc((void **)&hostFloats3, N * sizeof(float), hipHostMallocPortable);
    hipHostAlloc((void **)&hostFloats4, N * sizeof(float), hipHostMallocPortable);

    hipDeviceptr_t deviceFloats1;
    hipDeviceptr_t deviceFloats2;
    hipDeviceptr_t deviceFloats3;
    hipDeviceptr_t deviceFloats4;
    hipMalloc(&deviceFloats1, N * sizeof(float));
    hipMalloc(&deviceFloats2, N * sizeof(float));
    hipMalloc(&deviceFloats3, N * sizeof(float));
    hipMalloc(&deviceFloats4, N * sizeof(float));

    MyStruct2 mystruct;
    mystruct.floats1 = (float *)deviceFloats1;
    mystruct.floats2 = (float *)deviceFloats2;
    hostFloats1[0] = 123;
    hostFloats2[0] = 333;

    hipMemcpyHtoDAsync(
        (hipDeviceptr_t)(((float *)deviceFloats1)),
        hostFloats1,
        N * sizeof(float),
        stream
    );
    hipMemcpyHtoDAsync(
        (hipDeviceptr_t)(((float *)deviceFloats2)),
        hostFloats2,
        N * sizeof(float),
        stream
    );

    getValue2<<<dim3(1,1,1), dim3(32,1,1), 0, stream>>>(mystruct, (float *)deviceFloats3, (float *)deviceFloats4);

    hipMemcpyDtoHAsync(hostFloats3, deviceFloats3, N * sizeof(float), stream);
    hipMemcpyDtoHAsync(hostFloats4, deviceFloats4, N * sizeof(float), stream);

    hipStreamSynchronize(stream);

    // and check the values...
    cout << "hostFloats3[0] " << hostFloats3[0] << endl;
    cout << "hostFloats4[0] " << hostFloats4[0] << endl;

    assert(hostFloats3[0] == 126);
    assert(hostFloats4[0] == 338);

    hipHostFree(hostFloats1);
    hipHostFree(hostFloats2);
    hipHostFree(hostFloats3);
    hipHostFree(hostFloats4);
    hipFree(deviceFloats1);
    hipFree(deviceFloats2);
    hipFree(deviceFloats3);
    hipFree(deviceFloats4);
}

int main(int argc, char *argv[]) {
    hipStream_t stream;
    hipStreamCreateWithFlags(&stream, 0);

    // test1(stream);
    test2(stream);

    hipStreamDestroy(stream);
    return 0;
}
