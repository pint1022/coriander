
#include <iostream>
#include <memory>
#include <cassert>

using namespace std;

#include <hip/hip_runtime.h>
#include "hipblas.h"

void transpose(float *in, float *out, int rows, int cols) {
    int newrows = cols;
    int newcols = rows;
    for(int oldrow = 0; oldrow < rows; oldrow++) {
        for(int oldcol = 0; oldcol < cols; oldcol++) {
            int newcol = oldrow;
            int newrow = oldcol;
            out[newrow * newcols + newcol] = in[oldrow * cols + oldcol];
        }
    }
}

void dump(float *M, int rows, int cols) {
    for(int row=0; row < rows; row++) {
        for(int col = 0; col < cols; col++) {
            cout << " " << M[row * cols + col];
        }
        cout << endl;
    }
}

void fill(float *M, int rows, int cols, float val) {
    for(int row=0; row < rows; row++) {
        for(int col = 0; col < cols; col++) {
            M[row * cols + col] = val;
        }
    }
}

void dumbMultiply(float *A, float *B, float *C, int M, int N, int K) {
    // assume row major
    for(int m=0; m < M; m++) {
        for(int n = 0; n < N; n++) {
            float sum = 0;
            for(int k=0; k < K; k++) {
                sum += A[m * K + k] * B[k * N + n];
            }
            C[m * N + n] = sum;
        }
    }
}

void assertEqual(float *left, float *right, int rows, int cols) {
    for(int row=0; row < rows; row++) {
        for(int col=0; col < cols; col++) {
            if(left[row * cols + col] != right[row * cols + col]) {
                cout << "mismatch row=" << row << " col=" << col << " " << left[row * cols + col]
                    << " != " << right[row * cols + col] << endl;
                assert(left[row * cols + col] == right[row * cols + col]);
            }
        }
    }
}

int main(int argc, char *argv[]) {
    const int M = 3;
    const int K = 2;
    const int N = 4;

    hipStream_t stream;
    hipStreamCreateWithFlags(&stream, 0);

    float hostA[] = { 3, 5,
                      5, 8,
                      2, -1};
    float hostB[] = { 3, 5, 4, 1,
                      5, 8, 5, 7};
    float hostATrans[M * K];
    float hostBTrans[K * N];
    float hostC[M * N];
    float hostCTrans[M * N];

    transpose(hostA, hostATrans, M, K);
    transpose(hostB, hostBTrans, K, N);

    cout << "A:" << endl;
    dump(hostA, M, K);
    cout << "B:" << endl;
    dump(hostB, K, N);

    cout << "ATrans:" << endl;
    dump(hostATrans, K, M);
    cout << "BTrans:" << endl;
    dump(hostBTrans, N, K);

    hipDeviceptr_t deviceA;
    hipDeviceptr_t deviceB;
    hipDeviceptr_t deviceC;
    hipMalloc(&deviceA, M * K * sizeof(float) + 1024 * 4);
    hipMalloc(&deviceB, K * N * sizeof(float) + 1024 * 4);
    hipMalloc(&deviceC, M * N * sizeof(float) + 1024 * 4);

    int A_offset = 64;
    int B_offset = 128;
    int C_offset = 32;

    A_offset = 0;
    B_offset = 0;
    C_offset = 0;

    hipMemcpyHtoDAsync((hipDeviceptr_t)(((float *)deviceA) + A_offset), hostATrans, M * K * sizeof(float), stream);
    hipMemcpyHtoDAsync((hipDeviceptr_t)(((float *)deviceB) + B_offset), hostBTrans, K * N * sizeof(float), stream);

    hipblasHandle_t blas;
    hipblasCreate(&blas);
    // cout << "blas " << (void *)blas << endl;
    hipblasSetStream(blas, stream);

    float alpha = 1;
    float beta = 0;
      hipblasSgemm(blas, HIPBLAS_OP_N, HIPBLAS_OP_N, M, N, K,
      &alpha,
      (float *)deviceA + A_offset, M, (float *)deviceB + B_offset, K, &beta, (float *)deviceC + C_offset, M);

    hipMemcpyDtoHAsync(hostCTrans, (hipDeviceptr_t)((float *)deviceC + C_offset), M * N * sizeof(float), stream);
    hipStreamSynchronize(stream);

    cout << "C trans:" << endl;
    dump(hostCTrans, N, M);
    transpose(hostCTrans, hostC, N, M);
    cout << "C:" << endl;
    dump(hostC, M, N);

    float hostCCheck[M * N];
    dumbMultiply(hostA, hostB, hostCCheck, M, N, K);
    cout << "C check:" << endl;
    dump(hostCCheck, M, N);

    assertEqual(hostCCheck, hostC, M, N);

    // cublasDestroy(blas);

    hipFree(deviceA);
    hipFree(deviceB);
    hipFree(deviceC);
    hipStreamDestroy(stream);
    cout << "finished testblas" << endl;

    return 0;
}
