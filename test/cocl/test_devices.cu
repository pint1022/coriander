// based (very) loosely on cudnn-training's lenet.cu, and tensorflow's cl_driver.cc

#include <hip/hip_runtime.h>
#include <iostream>

int main(int argc, char *argv[]) {
    int num_gpus;
    hipGetDeviceCount(&num_gpus);
    std::cout << "num gpus: " << num_gpus << std::endl;

    for(int gpu=0; gpu < num_gpus; gpu++) {
        hipSetDevice(gpu);
        hipDevice_t device;
        hipGetDevice(&device);
        std::cout << "device: " << device << std::endl;
    }

    for(int gpu=0; gpu < num_gpus; gpu++) {
        hipCtx_t context;
        hipCtxCreate(&context, 0, gpu);
    }

    return 0;
}
