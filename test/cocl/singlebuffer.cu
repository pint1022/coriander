
#include <iostream>
#include <memory>
#include <cassert>

using namespace std;

#include <hip/hip_runtime.h>

__global__ void getValue(float *outdata, float *indata) {
    outdata[0] = indata[0] + 3.0f;
}

void floats() {
    int memSizeBytes = 65536;
    int N = 128;

    hipStream_t stream;
    hipStreamCreateWithFlags(&stream, 0);

    char *hostMemGiant;
    hipHostAlloc((void **)&hostMemGiant, memSizeBytes, hipHostMallocPortable);

    hipDeviceptr_t deviceMemGiant;
    hipMalloc(&deviceMemGiant, memSizeBytes);

    int floats1_offset_bytes = 128 * 4;
    int floats2_offset_bytes = 256 * 4;

    float *hostFloats1 = (float *)(hostMemGiant + floats1_offset_bytes);
    float *hostFloats2 = (float *)(hostMemGiant + floats2_offset_bytes);

    hostFloats1[0] = 123.456f;

    hipDeviceptr_t deviceFloats1 = deviceMemGiant + floats1_offset_bytes;
    hipDeviceptr_t deviceFloats2 = deviceMemGiant + floats2_offset_bytes;

    hipMemcpyHtoDAsync(
        (hipDeviceptr_t)(((float *)deviceFloats1)),
        hostFloats1,
        N * sizeof(float),
        stream
    );

    getValue<<<dim3(1,1,1), dim3(32,1,1), 0, stream>>>((float *)deviceFloats2, (float *)deviceFloats1);

    // now copy back entire buffer
    hipMemcpyDtoHAsync(hostFloats2, deviceFloats2, N * sizeof(float), stream);
    hipStreamSynchronize(stream);

    // and check the values...
    cout << hostFloats2[0] << endl;

    assert(hostFloats2[0] == 126.456f);

    hipHostFree(hostMemGiant);
    hipFree(deviceMemGiant);
    hipStreamDestroy(stream);
}

__global__ void getValueChar(char *outdata, char *indata) {
    outdata[0] = indata[0] + 3;
}

void chars() {
    int memSizeBytes = 65536;
    int N = 128;

    hipStream_t stream;
    hipStreamCreateWithFlags(&stream, 0);

    char *hostMemGiant;
    hipHostAlloc((void **)&hostMemGiant, memSizeBytes, hipHostMallocPortable);

    hipDeviceptr_t deviceMemGiant;
    hipMalloc(&deviceMemGiant, memSizeBytes);

    int chars1_offset_bytes = 128 * 4;
    int chars2_offset_bytes = 256 * 4;

    char *hostChars1 = (char *)(hostMemGiant + chars1_offset_bytes);
    char *hostChars2 = (char *)(hostMemGiant + chars2_offset_bytes);

    hostChars1[0] = 67;

    hipDeviceptr_t deviceChars1 = deviceMemGiant + chars1_offset_bytes;
    hipDeviceptr_t deviceChars2 = deviceMemGiant + chars2_offset_bytes;

    hipMemcpyHtoDAsync(
        (hipDeviceptr_t)(((float *)deviceChars1)),
        hostChars1,
        N * sizeof(char),
        stream
    );

    getValueChar<<<dim3(1,1,1), dim3(32,1,1), 0, stream>>>((char *)deviceChars2, (char *)deviceChars1);

    // now copy back entire buffer
    hipMemcpyDtoHAsync(hostChars2, deviceChars2, N * sizeof(char), stream);
    hipStreamSynchronize(stream);

    // and check the values...
    cout << hostChars2[0] << endl;

    assert(hostChars2[0] == 70);

    hipHostFree(hostMemGiant);
    hipFree(deviceMemGiant);
    hipStreamDestroy(stream);
}

int main(int argc, char *argv[]) {
    floats();
    chars();
    return 0;
}
