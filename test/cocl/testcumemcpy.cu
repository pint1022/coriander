// a cuda app.  we will convert this to opencl, and run it :-)

#include <iostream>
#include <memory>
#include <cassert>

using namespace std;

#include <hip/hip_runtime.h>

__global__ void setValue(float *data, int idx, float value) {
    if(threadIdx.x == 0) {
        data[idx] = value;
    }
}

int main(int argc, char *argv[]) {
    int N = 1024;

    hipDeviceptr_t gpuFloats;
    hipMalloc(&gpuFloats, N * sizeof(float));

    setValue<<<dim3(32, 1, 1), dim3(32, 1, 1)>>>((float *)gpuFloats, 2, 123.0f);

    float hostFloats[4];
    hipMemcpyDtoH(hostFloats, gpuFloats, 4 * sizeof(float));
    cout << "hostFloats[2] " << hostFloats[2] << endl;
    assert(hostFloats[2] == 123);

    setValue<<<dim3(32, 1, 1), dim3(32, 1, 1)>>>((float *)gpuFloats, 2, 222.0f);
    hipMemcpyDtoH(hostFloats, gpuFloats, 4 * sizeof(float));
    cout << "hostFloats[2] " << hostFloats[2] << endl;
    assert(hostFloats[2] == 222);

    hostFloats[2] = 444.0f;
    hipMemcpyHtoD(gpuFloats, hostFloats, 4 * sizeof(float));
    hostFloats[2] = 555.0f;
    hipMemcpyDtoH(hostFloats, gpuFloats, 4 * sizeof(float));
    cout << "hostFloats[2] " << hostFloats[2] << endl;
    assert(hostFloats[2] == 444);

    hipFree(gpuFloats);

    return 0;
}
