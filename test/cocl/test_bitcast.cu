// seems my bitcasting is/was broken ('is', at time of writing this test, 'was' since I probably fixed it by now :-) )
// this code tests this, sortof
// (hmmm, edit, seems to be ok, in fact...)

#include "hip/hip_runtime.h"

#include <iostream>
#include <cassert>

using namespace std;

__global__ void mykernel(int *int1, float *f1, int *int2) {
    f1[0] = *(float *)&int1[0];
    int2[0] = *(int *)&f1[0];
}

__global__ void floattoint(int *out, float *in) {
    out[0] = *(int *)&in[0];
}

__global__ void inttofloat(float *out, int *in) {
    out[0] = *(float *)&in[0];
}

int main(int argc, char *argv[]) {

    int N = 32;
    int *int1_host = new int[N];
    float *f1_host = new float[N];
    int *int2_host = new int[N];
    // float *f2_host = new float[N];

    int *int1_gpu;
    float *f1_gpu;
    int *int2_gpu;
    // float *f2_gpu;
    hipMalloc((void**)&int1_gpu, N * sizeof(float));
    hipMalloc((void**)&f1_gpu, N * sizeof(float));
    hipMalloc((void**)&int2_gpu, N * sizeof(float));
    // cudaMalloc(&(void**)f2_gpu, N * sizeof(float));

    int1_host[0] = 123;
    hipMemcpy(int1_gpu, int1_host, N * sizeof(float), hipMemcpyHostToDevice);

    mykernel<<<dim3(1,1,1), dim3(32,1,1)>>>(int1_gpu, f1_gpu, int2_gpu);

    hipMemcpy(f1_host, f1_gpu, N * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(int2_host, int2_gpu, N * sizeof(float), hipMemcpyDeviceToHost);
    cout << "this should NOT be 123, should be some weird float value, not even slightly close to 123 :" << endl;
    cout << "f1[0] " << f1_host[0] << endl;
    assert(f1_host[0] != 123.0f);
    cout << "this SHOULD be 123 :" << endl;
    cout << "int2[0] " << int2_host[0] << endl;
    assert(int2_host[0] == 123);

    int1_host[0] = 777;
    hipMemcpy(int1_gpu, int1_host, N * sizeof(float), hipMemcpyHostToDevice);
    cout << "after copy to device" << endl;
    inttofloat<<<dim3(1,1,1), dim3(32,1,1)>>>(f1_gpu, int1_gpu);
    cout << "after kernel call 2" << endl;
    hipMemcpy(f1_host, f1_gpu, N * sizeof(float), hipMemcpyDeviceToHost);
    cout << "f1[0]" << f1_host[0] << endl;
    assert(f1_host[0] != 777.0f);

    floattoint<<<dim3(1,1,1), dim3(32,1,1)>>>(int2_gpu, f1_gpu);
    cout << "after kernel call 3" << endl;
    hipMemcpy(int2_host, int2_gpu, N * sizeof(float), hipMemcpyDeviceToHost);
    cout << "int2[0]" << int2_host[0] << endl;
    assert(int2_host[0] == 777);

    return 0;
}
