// a cuda app.  we will convert this to opencl, and run it :-)

#include <iostream>
#include <memory>
#include <cassert>

using namespace std;

#include <hip/hip_runtime.h>

__global__ void setValue(char *data, int idx, char value) {
    if(threadIdx.x == 0) {
        data[idx] = value;
    }
}

int main(int argc, char *argv[]) {
    int N = 1024;

    char *gpuChars;
    hipMalloc((void**)(&gpuChars), N * sizeof(char));

    setValue<<<dim3(32, 1, 1), dim3(32, 1, 1)>>>(gpuChars, 2, 89);

    char hostChars[4];
    hipMemcpy(hostChars, gpuChars, 4 * sizeof(char), hipMemcpyDeviceToHost);
    cout << "hostChars[2] " << hostChars[2] << endl;
    assert(hostChars[2] == 89);

    setValue<<<dim3(32, 1, 1), dim3(32, 1, 1)>>>(gpuChars, 2, 23);
    hipMemcpy(hostChars, gpuChars, 4 * sizeof(char), hipMemcpyDeviceToHost);
    cout << "hostChars[2] " << hostChars[2] << endl;
    assert(hostChars[2] == 23);

    hostChars[2] = 44;
    hipMemcpy(gpuChars, hostChars, 4 * sizeof(char), hipMemcpyHostToDevice);
    hostChars[2] = 55;
    hipMemcpy(hostChars, gpuChars, 4 * sizeof(char), hipMemcpyDeviceToHost);
    cout << "hostChars[2] " << hostChars[2] << endl;
    assert(hostChars[2] == 44);

    hipFree(gpuChars);

    return 0;
}
