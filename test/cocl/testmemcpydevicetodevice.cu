// a cuda app.  we will convert this to opencl, and run it :-)

#include <iostream>
#include <memory>
#include <cassert>

using namespace std;

#include <hip/hip_runtime.h>

__global__ void setValue(float *data, int idx, float value) {
    if(threadIdx.x == 0) {
        data[idx] = value;
    }
}

int main(int argc, char *argv[]) {
    int N = 1024;

    float *gpuFloats1;
    hipMalloc((void**)(&gpuFloats1), N * sizeof(float));

    float *gpuFloats2;
    hipMalloc((void**)(&gpuFloats2), N * sizeof(float));

    setValue<<<dim3(32, 1, 1), dim3(32, 1, 1)>>>(gpuFloats1, 2, 123.0f);
    hipMemcpy(gpuFloats2, gpuFloats1, 4 * sizeof(float), hipMemcpyDeviceToDevice);

    setValue<<<dim3(32, 1, 1), dim3(32, 1, 1)>>>(gpuFloats1, 2, 444.0f);

    float hostFloats[4];
    hipMemcpy(hostFloats, gpuFloats2, 4 * sizeof(float), hipMemcpyDeviceToHost);
    cout << "This should be 123:" << endl;
    cout << "hostFloats[2] " << hostFloats[2] << endl;
    assert(hostFloats[2] == 123);

    hipFree(gpuFloats1);

    return 0;
}
