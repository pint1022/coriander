#include "hip/hip_runtime.h"
#include "multi1.h"

#include <iostream>
using namespace std;

__global__ void main_set(float *data, float val) {
    data[threadIdx.x] = val;
}

int main(int argc, char *argv[]) {
    cout << "main" << endl;

    int N = 1024;

    hipStream_t newstream;
    hipStreamCreateWithFlags(&newstream, 0);
    cout << "got stream" << endl;

    float *hostfloats;
    hipHostAlloc((void **)&hostfloats, N * sizeof(float), hipHostMallocPortable);

    hipDeviceptr_t devicefloats;
    hipMalloc(&devicefloats, N * sizeof(float));

    main_set<<<dim3(1, 1, 1), dim3(32, 1, 1), 0, newstream>>>((float *)devicefloats, 5.0f);
    cout << "queued kernel 1" << endl;

    k1_incr<<<dim3(1, 1, 1), dim3(32, 1, 1), 0, newstream>>>((float *)devicefloats, 5.0f);
    cout << "queued kernel 1" << endl;

    k2_mul<<<dim3(11, 1, 1), dim3(32, 1, 1), 0, newstream>>>((float *)devicefloats, 3.0f);
    cout << "queued kernel 2" << endl;


    hipMemcpyDtoHAsync(hostfloats, devicefloats, N * sizeof(float), newstream);
    hipStreamSynchronize(newstream);

    cout << "hostFloats[2] " << hostfloats[2] << endl;

    cout << "finished" << endl;

    hipHostFree(hostfloats);
    hipFree(devicefloats);
    hipStreamDestroy(newstream);

    cout << "done" << endl;

    return 0;
}
