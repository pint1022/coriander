#include "hip/hip_runtime.h"
#include "mylib.h"

#include <iostream>
#include <memory>
#include <cassert>

using namespace std;

int main(int argc, char *argv[]) {
    int N = 1024;

    hipStream_t stream;
    hipStreamCreateWithFlags(&stream, 0);

    float *hostFloats1;
    hipHostAlloc((void **)&hostFloats1, N * sizeof(float), hipHostMallocPortable);

    hipDeviceptr_t deviceFloats1;
    hipMalloc(&deviceFloats1, N * sizeof(float));

    hostFloats1[0] = 123.456f;

    hipMemcpyHtoDAsync(
        (hipDeviceptr_t)(((float *)deviceFloats1)),
        hostFloats1,
        N * sizeof(float),
        stream
    );

    __global__ void mykernel(float *data);
    mykernel<<<dim3(1,1,1), dim3(32,1,1), 0, stream>>>(((float *)deviceFloats1));

    hipMemcpyDtoHAsync(hostFloats1, deviceFloats1, N * sizeof(float), stream);
    hipStreamSynchronize(stream);

    cout << hostFloats1[0] << endl;

    assert(hostFloats1[0] == 126.456f);

    hipHostFree(hostFloats1);
    hipFree(deviceFloats1);
    hipStreamDestroy(stream);

    return 0;
}
