
#include <iostream>
#include <memory>
#include <cassert>

using namespace std;

#include <hip/hip_runtime.h>

__global__ void getValue(float4 *outdata, float *indata) {
    // outdata[0] = indata[0];
    float4 my4 = make_float4(indata[0], indata[3], indata[1], indata[2]);
    outdata[0] = my4;
}

int main(int argc, char *argv[]) {
    int N = 1024;

    hipStream_t stream;
    hipStreamCreateWithFlags(&stream, 0);

    float *hostFloats1;
    float *hostFloats2;
    hipHostAlloc((void **)&hostFloats1, N * sizeof(float), hipHostMallocPortable);
    hipHostAlloc((void **)&hostFloats2, N * sizeof(float), hipHostMallocPortable);

    hipDeviceptr_t deviceFloats1;
    hipDeviceptr_t deviceFloats2;
    hipMalloc(&deviceFloats1, N * sizeof(float));
    hipMalloc(&deviceFloats2, N * sizeof(float));

    hostFloats1[0] = 123.456f;
    hostFloats1[1] = 9;
    hostFloats1[2] = 12;
    hostFloats1[3] = 19;

    hipMemcpyHtoDAsync(
        (hipDeviceptr_t)(((float *)deviceFloats1)),
        hostFloats1,
        N * sizeof(float),
        stream
    );
    // cuStreamSynchronize(stream);

    getValue<<<dim3(1,1,1), dim3(32,1,1), 0, stream>>>(((float4 *)deviceFloats2), ((float *)deviceFloats1));

    // now copy back entire buffer
    // hostFloats[64] = 0.0f;
    hipMemcpyDtoHAsync(hostFloats2, deviceFloats2, N * sizeof(float), stream);
    hipStreamSynchronize(stream);

    // and check the values...
    cout << hostFloats2[0] << endl;
    cout << hostFloats2[1] << endl;
    cout << hostFloats2[2] << endl;
    cout << hostFloats2[3] << endl;

    // assert(hostFloats2[64] == 126.456f);

    hipHostFree(hostFloats1);
    hipHostFree(hostFloats2);
    hipFree(deviceFloats1);
    hipFree(deviceFloats2);
    hipStreamDestroy(stream);

    return 0;
}
