// What if we are given a device pointer that is offset from any of the device pointers we provided to the client?
//
// This file is a test-case for this.  Then we can look at handling that...

#include <iostream>
#include <memory>
#include <cassert>

using namespace std;

#include <hip/hip_runtime.h>

// __global__ void getValue(float *data, int idx, float value) {
//     if(threadIdx.x == 0) {
//         data[idx] += value;
//     }
// }

int main(int argc, char *argv[]) {
    int N = 1024;

    hipStream_t stream;
    hipStreamCreateWithFlags(&stream, 0);

    float *hostFloats;
    hipHostAlloc((void **)&hostFloats, N * sizeof(float), hipHostMallocPortable);

    hipDeviceptr_t deviceFloats;
    hipMalloc(&deviceFloats, N * sizeof(float));

    hostFloats[128] = 123.456f;
    hostFloats[129] = 444.0f;
    hostFloats[130] = 321.0f;
    hostFloats[131] = 111.0f;

    // now we will copy 16 bytes, starting at location 128...
    hipMemcpyHtoDAsync(
        (hipDeviceptr_t)(((float *)deviceFloats) + 64),
        hostFloats + 128,
        4 * sizeof(float),
        stream
    );
    hipStreamSynchronize(stream);
    // now copy back entire buffer
    hostFloats[64] = 0.0f;
    hostFloats[65] = 0.0f;
    hostFloats[66] = 0.0f;
    hostFloats[67] = 0.0f;
    hipMemcpyDtoHAsync(hostFloats, deviceFloats, N * sizeof(float), stream);
    hipStreamSynchronize(stream);

    // and check the values...
    cout << hostFloats[64] << endl;
    cout << hostFloats[65] << endl;
    cout << hostFloats[66] << endl;
    cout << hostFloats[67] << endl;

    assert(hostFloats[64] == 123.456f);
    assert(hostFloats[65] == 444.0f);
    assert(hostFloats[66] == 321);
    assert(hostFloats[67] == 111);

    hipHostFree(hostFloats);
    hipFree(deviceFloats);
    hipStreamDestroy(stream);

    return 0;
}
