// tests cuMemHostAlloc

#include <iostream>
#include <memory>
#include <cassert>

using namespace std;

#include <hip/hip_runtime.h>

__global__ void incrValue(float *data, int idx, float value) {
    if(threadIdx.x == 0  && blockIdx.x == 0) {
        data[idx] += value;
    }
}

int main(int argc, char *argv[]) {
    int N = 1024;

    hipStream_t stream;
    hipStreamCreateWithFlags(&stream, 0);

    float *hostFloats;
    hipHostAlloc((void **)&hostFloats, N * sizeof(float), hipHostMallocPortable);

    hipDeviceptr_t deviceFloats;
    hipMalloc(&deviceFloats, N * sizeof(float));

    hostFloats[2] = 4.0f;
    hipMemcpyHtoDAsync(deviceFloats, hostFloats, N * sizeof(float), stream);
    incrValue<<<dim3(32, 1, 1), dim3(32, 1, 1), 0, stream>>>((float *)deviceFloats, 2, 3.0f);
    hipMemcpyDtoHAsync(hostFloats, deviceFloats, N * sizeof(float), stream);
    hipStreamSynchronize(stream);
    cout << "hostFloats[2] " << hostFloats[2] << endl;
    assert(hostFloats[2] == 7);

    incrValue<<<dim3(32, 1, 1), dim3(32, 1, 1), 0, stream>>>((float *)deviceFloats, 2, 5.0f);
    hipMemcpyDtoHAsync(hostFloats, deviceFloats, N * sizeof(float), stream);
    hipStreamSynchronize(stream);
    cout << "hostFloats[2] " << hostFloats[2] << endl;
    assert(hostFloats[2] == 12);

    hipHostFree(hostFloats);
    hipFree(deviceFloats);
    hipStreamDestroy(stream);

    return 0;
}
