
#include <iostream>
#include <memory>
#include <cassert>

using namespace std;

#include <hip/hip_runtime.h>

__global__ void getValue(float *data) {
    data[0] = pow(data[1], data[2]);
    data[4] = min(data[1], data[2]);
    data[5] = max(data[1], data[2]);
    data[6] = ::max(data[1], data[2]);
    // data[7] = std::max(data[1], data[2]);
}

int main(int argc, char *argv[]) {
    int N = 1024;

    hipStream_t stream;
    hipStreamCreateWithFlags(&stream, 0);

    float *hostFloats1;
    hipHostAlloc((void **)&hostFloats1, N * sizeof(float), hipHostMallocPortable);

    hipDeviceptr_t deviceFloats1;
    hipMalloc(&deviceFloats1, N * sizeof(float));

    hostFloats1[0] = 0;
    hostFloats1[1] = 3;
    hostFloats1[2] = 4.5f;

    hipMemcpyHtoDAsync(
        (hipDeviceptr_t)(((float *)deviceFloats1)),
        hostFloats1,
        N * sizeof(float),
        stream
    );
    // cuStreamSynchronize(stream);

    getValue<<<dim3(1,1,1), dim3(32,1,1), 0, stream>>>(((float *)deviceFloats1));

    hipMemcpyDtoHAsync(hostFloats1, deviceFloats1, N * sizeof(float), stream);
    hipStreamSynchronize(stream);

    // and check the values...
    for(int i = 0; i < 7; i++) {
        cout << "hostFloats1[" << i << "]=" << hostFloats1[i] << endl;
    }
    // cout << hostFloats1[0] << endl;
    // cout << hostFloats1[1] << endl;
    // cout << hostFloats1[2] << endl;

    // cout << hostFloats1[4] << endl;
    // cout << hostFloats1[5] << endl;

    float diff = std::abs(hostFloats1[0] - 140.296);
    assert(diff < 0.01);

    assert(hostFloats1[4] == 3);
    assert(hostFloats1[5] == 4.5f);

    hipHostFree(hostFloats1);
    hipFree(deviceFloats1);
    hipStreamDestroy(stream);

    return 0;
}
