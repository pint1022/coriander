#include "hip/hip_runtime.h"
// test calling kernels from different threads, in parallel (can be different kernels, or same.  either way, should work, not crash :-) )

#include "pthread.h"

#include "hostside_opencl_funcs.h"

#include <iostream>
#include <memory>
#include <cassert>
#include <sstream>

using namespace std;

#include <hip/hip_runtime.h>

const int N = 1024;

__global__ void getValue(float *outdata, float *indata) {
    outdata[0] = indata == 0 ? 3.0f : 2.0f;
}

template<typename T>
static std::string toString(T val) {
   std::ostringstream myostringstream;
   myostringstream << val;
   return myostringstream.str();
}

pthread_mutex_t print_mutex = PTHREAD_MUTEX_INITIALIZER;
void print(string message) {
    pthread_mutex_lock(&print_mutex);
    cout << message << endl;
    pthread_mutex_unlock(&print_mutex);
}

void *thread_func(void *data) {
    int i = (size_t)data;
    print("thread " + toString(i));
    hipStream_t stream;
    hipStreamCreateWithFlags(&stream, 0);

    float *hostFloats1;
    hipHostAlloc((void **)&hostFloats1, N * sizeof(float), hipHostMallocPortable);

    hipDeviceptr_t deviceFloats1;
    hipMalloc(&deviceFloats1, N * sizeof(float));

    getValue<<<dim3(1,1,1), dim3(32,1,1), 0, stream>>>(((float *)deviceFloats1), 0);
    getValue<<<dim3(1,1,1), dim3(32,1,1), 0, stream>>>(((float *)deviceFloats1), 0);
    getValue<<<dim3(1,1,1), dim3(32,1,1), 0, stream>>>(((float *)deviceFloats1), 0);
    getValue<<<dim3(1,1,1), dim3(32,1,1), 0, stream>>>(((float *)deviceFloats1), 0);
    hipStreamSynchronize(stream);

    print("num kernels cached " + toString(cocl::getNumCachedKernels()));
    print("num kernels calls " + toString(cocl::getNumKernelCalls()));

    assert(cocl::getNumCachedKernels() == 1);
    assert(cocl::getNumKernelCalls() == 4);

    hipHostFree(hostFloats1);
    hipFree(deviceFloats1);
    hipStreamDestroy(stream);
    return 0;
}

void testfloatstar() {
    const int NUM_THREADS = 4;
    pthread_t threads[ NUM_THREADS ];
    for(long long i = 0; i < NUM_THREADS; i++) {
        pthread_create(&threads[i], NULL, thread_func, (void *)i);
    }
    cout << "creaed threads" << endl;
    for(int i = 0; i < NUM_THREADS; i++) {
        pthread_join(threads[i], NULL);
        cout << "joined thread " << i << endl;
    }
}

int main(int argc, char *argv[]) {
    testfloatstar();
    return 0;
}
