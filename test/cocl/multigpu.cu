#include "hip/hip_runtime.h"
// test calling kernels from different threads, in parallel (can be different kernels, or same.  either way, should work, not crash :-) )

#include "pthread.h"

#include "hostside_opencl_funcs.h"

#include <iostream>
#include <memory>
#include <cassert>
#include <sstream>

using namespace std;

#include <hip/hip_runtime.h>

const int N = 1024;

__global__ void getValue(float *outdata) {
    float sum = 0;
    for(int i = 1; i < N; i++) {
        sum += outdata[i];
    }
    outdata[0] = sum;
}

template<typename T>
static std::string toString(T val) {
   std::ostringstream myostringstream;
   myostringstream << val;
   return myostringstream.str();
}

pthread_mutex_t print_mutex = PTHREAD_MUTEX_INITIALIZER;
void print(string message) {
    pthread_mutex_lock(&print_mutex);
    cout << message << endl;
    pthread_mutex_unlock(&print_mutex);
}

// hipCtx_t context;
hipCtx_t *context;

pthread_mutex_t context_mutex = PTHREAD_MUTEX_INITIALIZER;

void *thread_func(void *data) {
    int i = (size_t)data;
    print("thread " + toString(i));

    hipCtxSetCurrent(context[i]);

    float *hostFloats1;
    hipDeviceptr_t deviceFloats1;

    hipHostAlloc((void **)&hostFloats1, N * sizeof(float), hipHostMallocPortable);
    hipMalloc(&deviceFloats1, N * sizeof(float));

    hipStream_t stream;
    hipStreamCreateWithFlags(&stream, 0);

    for(int it=0; it < 10; it++) {
        getValue<<<dim3(1,1,1), dim3(32,1,1), 0, stream>>>(((float *)deviceFloats1));
    }

    hipStreamSynchronize(stream);

    hipHostFree(hostFloats1);
    hipFree(deviceFloats1);

    hipStreamDestroy(stream);
    return 0;
}

void testfloatstar() {
    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    cout << "devices: " << deviceCount << endl;
    context = new hipCtx_t[deviceCount];
    // int NUM_THREADS = 2;

    for(int i = 0; i < deviceCount; i++) {
        hipCtxCreate(&context[i], 0, i);
        cout << "created context " << (void *)context[i] << endl;
    }
    cout << "created contexts" << endl;

    pthread_t threads[ deviceCount ];
    for(long long i = 0; i < deviceCount; i++) {
        pthread_create(&threads[i], NULL, thread_func, (void *)i);
    }
    cout << "creaed threads" << endl;
    for(int i = 0; i < deviceCount; i++) {
        pthread_join(threads[i], NULL);
        cout << "joined thread " << i << endl;
    }

    print("num kernels cached " + toString(cocl::getNumCachedKernels()));
    print("num kernels calls " + toString(cocl::getNumKernelCalls()));

    for(int i = 0; i < deviceCount; i++) {
        hipCtxSetCurrent(context[i]);
        assert(cocl::getNumCachedKernels() == 1);
        assert(cocl::getNumKernelCalls() == 10);
    }

    delete[] context;
}

int main(int argc, char *argv[]) {
    testfloatstar();
    return 0;
}
