#include "hip/hip_runtime.h"
// tests hipEventCreateWithFlags

#include <iostream>
#include <memory>
#include <unistd.h>

using namespace std;

#include <hip/hip_runtime.h>

__global__ void longKernel(float *data, int N, float value) {
    for(int i = 0; i < N; i++) {
        data[i] += value;
    }
}

void myCallback(hipStream_t stream, size_t status, void *data) {
    char *message = (char *)data;
    cout << "message " << message << endl;
}

int main(int argc, char *argv[]) {
    int N = 52400; // * 1024;

    float *hostfloats = new float[N];
    float *gpufloats;
    hipMalloc((void **)&gpufloats, N * sizeof(float));

    hipStream_t stream;
    hipStreamCreateWithFlags(&stream, 0);

    longKernel<<<dim3(102400 / 32, 1, 1), dim3(32, 1, 1), 0, stream>>>(gpufloats, N, 3.0f);
    cout << "queued kernel x" << endl;

    const char *message = "hello";
    hipStreamAddCallback(stream, myCallback, (void *)message, 0);
    cout << "added callback" << endl;
    sleep(1);

    cout << "synchronizing..." << endl;
    hipStreamSynchronize(stream);
    cout << "... synchronized" << endl;

    hipStreamDestroy(stream);
    hipFree(gpufloats);

    cout << "finished" << endl;

    return 0;
}
