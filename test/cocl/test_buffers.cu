#include <iostream>
#include <hip/hip_runtime.h>

__global__ void mykernel(float *d1, float *d2, float *d3, float *d4, float *d5) {
    if(threadIdx.x == 0) {
        d1[0] = 123.0f;
        d2[0] = 123.0f;
        d3[0] = 123.0f;
        d4[0] = 123.0f;
        d5[0] = 123.0f;
    }
}

int main(int argc, char *argv[]) {
    const int bufferSizeMegs = 512;
    const int bufferSize = bufferSizeMegs * 1024 * 1024;

    float *gpuFloats;
    hipMalloc((void**)(&gpuFloats), bufferSize);

    for(int i = 0; i < 1000; i++) {
        if(i % 100 == 0 || i == 999) {
            std::cout << "i=" << i << std::endl;
        }
        mykernel<<<dim3(1024, 1, 1), dim3(256, 1, 1)>>>(gpuFloats, gpuFloats, gpuFloats, gpuFloats, gpuFloats);
    }

    hipFree(gpuFloats);

    return 0;
}
