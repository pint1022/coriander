#include "hip/hip_runtime.h"
#include "test_callinternal.h"

#include <iostream>
#include <cassert>

__global__ void mykernel(MyStruct mystruct, float *data) {
    somefunc(mystruct);
    // mystruct.somefloats[0] += 4;
    // addfive(mystruct.somefloats);
    data[0] = mystruct.somefloats[0] * 2.0f;
}

namespace cocl {
    int getNumClSources();
}

int main(int argc, char *argv[]) {

    int N = 32;

    std::cout << "clSources.size() " << cocl::getNumClSources() << std::endl;

    float *data_host = new float[N];
    float *struct_floats_host = new float[N];

    float *data_gpu;
    float *struct_floats_gpu;

    hipMalloc((void**)&data_gpu, N * sizeof(float));
    hipMalloc((void**)&struct_floats_gpu, N * sizeof(float));

    struct_floats_host[0] = 123;
    hipMemcpy(struct_floats_gpu, struct_floats_host, N * sizeof(float), hipMemcpyHostToDevice);

    struct MyStruct mystruct;
    mystruct.somefloats = struct_floats_gpu;
    // hipMemcpy(gpu_floats, host_floats, N * sizeof(float), hipMemcpyHostToDevice);

    mykernel<<<dim3(1,1,1), dim3(32,1,1)>>>(mystruct, data_gpu);

    hipMemcpy(data_host, data_gpu, N * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(struct_floats_host, struct_floats_gpu, N * sizeof(float), hipMemcpyDeviceToHost);

    std::cout << "data_host[0]=" << data_host[0] << std::endl;
    std::cout << "struct_floats_host[0]=" << struct_floats_host[0] << std::endl;
    assert(data_host[0] == (123+3) * 2);

    return 0;
}

class MyClass {
public:
    MyClass(const char *name);
};

MyClass main_(__FILE__);
