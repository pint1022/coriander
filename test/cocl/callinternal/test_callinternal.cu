// I have/had a bug where internal function calls were processed to have struct arguments with pointers split out,
// just as for kernel calls
// This is a testcase to reproduce/fix/test that bug

#include "test_callinternal.h"

#include "hip/hip_runtime.h"

#include <iostream>
using namespace std;

__device__ void somefunc(struct MyStruct mystruct) {
    mystruct.somefloats[0] += 3.0f;
}

__device__ void hasstructpointer(struct MyStruct *mystruct) {
    mystruct[0].somefloats[0] += 3.0f;
}

__device__ void addfive(float *data) {
    data[0] += 5.0f;
}

class MyClass {
public:
    MyClass(const char *name);
};

MyClass callinternal(__FILE__);
