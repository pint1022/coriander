#include "hip/hip_runtime.h"
// for this one, we need some metadata on how many kernel compiles etc
// so we'll need to include some cocl include files, directly

#include "hostside_opencl_funcs.h"

#include <iostream>
#include <memory>
#include <cassert>

using namespace std;

#include <hip/hip_runtime.h>

__global__ void getValue(float *outdata, float *indata) {
    outdata[0] = indata == 0 ? 3.0f : 2.0f;
}

void testfloatstar() {
    int N = 1024;

    hipStream_t stream;
    hipStreamCreateWithFlags(&stream, 0);

    float *hostFloats1;
    hipHostAlloc((void **)&hostFloats1, N * sizeof(float), hipHostMallocPortable);

    hipDeviceptr_t deviceFloats1;
    hipMalloc(&deviceFloats1, N * sizeof(float));

    getValue<<<dim3(1,1,1), dim3(32,1,1), 0, stream>>>(((float *)deviceFloats1), 0);
    getValue<<<dim3(1,1,1), dim3(32,1,1), 0, stream>>>(((float *)deviceFloats1), 0);
    getValue<<<dim3(1,1,1), dim3(32,1,1), 0, stream>>>(((float *)deviceFloats1), 0);
    getValue<<<dim3(1,1,1), dim3(32,1,1), 0, stream>>>(((float *)deviceFloats1), 0);
    hipStreamSynchronize(stream);

    cout << "num kernels cached " << cocl::getNumCachedKernels() << endl;
    cout << "num kernel calls " << cocl::getNumKernelCalls() << endl;

    assert(cocl::getNumCachedKernels() == 1);
    assert(cocl::getNumKernelCalls() == 4);

    hipHostFree(hostFloats1);
    hipFree(deviceFloats1);
    hipStreamDestroy(stream);
}

int main(int argc, char *argv[]) {
    testfloatstar();
    return 0;
}

