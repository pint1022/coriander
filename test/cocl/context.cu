#include "hip/hip_runtime.h"
// test calling kernels from different threads, in parallel (can be different kernels, or same.  either way, should work, not crash :-) )

#include "pthread.h"

#include "hostside_opencl_funcs.h"

#include <iostream>
#include <memory>
#include <cassert>
#include <sstream>

using namespace std;

#include <hip/hip_runtime.h>

const int N = 1024;

__global__ void getValue(float *outdata) {
    float sum = 0;
    for(int i = 1; i < N; i++) {
        sum += outdata[i];
    }
    outdata[0] = sum;
}

template<typename T>
static std::string toString(T val) {
   std::ostringstream myostringstream;
   myostringstream << val;
   return myostringstream.str();
}

pthread_mutex_t print_mutex = PTHREAD_MUTEX_INITIALIZER;
void print(string message) {
    pthread_mutex_lock(&print_mutex);
    cout << message << endl;
    pthread_mutex_unlock(&print_mutex);
}

hipCtx_t context;
float *hostFloats1;
hipDeviceptr_t deviceFloats1;

pthread_mutex_t context_mutex = PTHREAD_MUTEX_INITIALIZER;

void *thread_func(void *data) {
    int i = (size_t)data;
    print("thread " + toString(i));

    hipCtxSetCurrent(context);

    hipStream_t stream;
    hipStreamCreateWithFlags(&stream, 0);

    for(int it=0; it < 10; it++) {
        getValue<<<dim3(1,1,1), dim3(32,1,1), 0, stream>>>(((float *)deviceFloats1));
    }

    hipStreamSynchronize(stream);
    hipStreamDestroy(stream);
    return 0;
}

void testfloatstar() {
    const int NUM_THREADS = 4;

    hipCtxCreate(&context, 0, 0);

    hipHostAlloc((void **)&hostFloats1, N * sizeof(float), hipHostMallocPortable);
    hipMalloc(&deviceFloats1, N * sizeof(float));

    pthread_t threads[ NUM_THREADS ];
    for(long long i = 0; i < NUM_THREADS; i++) {
        pthread_create(&threads[i], NULL, thread_func, (void *)i);
    }
    cout << "creaed threads" << endl;
    for(int i = 0; i < NUM_THREADS; i++) {
        pthread_join(threads[i], NULL);
        cout << "joined thread " << i << endl;
    }

    hipHostFree(hostFloats1);
    hipFree(deviceFloats1);

    print("num kernels cached " + toString(cocl::getNumCachedKernels()));
    print("num kernels calls " + toString(cocl::getNumKernelCalls()));

    assert(cocl::getNumCachedKernels() == 1);
    assert(cocl::getNumKernelCalls() == 40);
}

int main(int argc, char *argv[]) {
    testfloatstar();
    return 0;
}
