
#include <iostream>
#include <memory>
#include <cassert>

using namespace std;

#include <hip/hip_runtime.h>

__global__ void getValue(float *outdata, float *indata) {
    outdata[0] = indata == 0 ? 3.0f : 2.0f;
}

void testfloatstar() {
    int N = 1024;

    hipStream_t stream;
    hipStreamCreateWithFlags(&stream, 0);

    float *hostFloats1;
    hipHostAlloc((void **)&hostFloats1, N * sizeof(float), hipHostMallocPortable);

    hipDeviceptr_t deviceFloats1;
    hipMalloc(&deviceFloats1, N * sizeof(float));

    hipMemcpyHtoDAsync((hipDeviceptr_t)(((float *)deviceFloats1)), hostFloats1, N * sizeof(float), stream);
    getValue<<<dim3(1,1,1), dim3(32,1,1), 0, stream>>>(((float *)deviceFloats1), 0);
    hipMemcpyDtoHAsync(hostFloats1, deviceFloats1, N * sizeof(float), stream);
    hipStreamSynchronize(stream);

    cout << hostFloats1[0] << endl;

    assert(hostFloats1[0] == 3);

    hipHostFree(hostFloats1);
    hipFree(deviceFloats1);
    hipStreamDestroy(stream);
}

struct MyStruct {
    float *p1;
    float *p2;
};

__global__ void checkNullStructs(struct MyStruct mystruct) {
    mystruct.p1[0] = mystruct.p2 == 0 ? 9 : 8;
}

void teststruct() {
    int N = 1024;

    hipStream_t stream;
    hipStreamCreateWithFlags(&stream, 0);

    float *hostFloats1;
    hipHostAlloc((void **)&hostFloats1, N * sizeof(float), hipHostMallocPortable);

    hipDeviceptr_t deviceFloats1;
    hipMalloc(&deviceFloats1, N * sizeof(float));

    hipMemcpyHtoDAsync((hipDeviceptr_t)(((float *)deviceFloats1)), hostFloats1, N * sizeof(float), stream);
    struct MyStruct mystruct = {(float *)deviceFloats1, 0};
    checkNullStructs<<<dim3(1,1,1), dim3(32,1,1), 0, stream>>>(mystruct);
    hipMemcpyDtoHAsync(hostFloats1, deviceFloats1, N * sizeof(float), stream);
    hipStreamSynchronize(stream);

    cout << hostFloats1[0] << endl;

    assert(hostFloats1[0] == 9);

    hipHostFree(hostFloats1);
    hipFree(deviceFloats1);
    hipStreamDestroy(stream);
}

int main(int argc, char *argv[]) {
    testfloatstar();
    teststruct();
    return 0;
}
