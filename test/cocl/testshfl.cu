// test __shfl_down...

#include <iostream>
#include <memory>
#include <cassert>

using namespace std;

#include <hip/hip_runtime.h>

__global__ void getValue(float *data) {
    int tid = threadIdx.x;
    int warpid = tid % 32;  // assume warpsize 32.  Anyway, CUDA code uses warpsize 32.
    float me = data[tid];
    me = __shfl_down(me, 1, 32);
    data[tid] = me;
}

int main(int argc, char *argv[]) {
    int N = 1024;

    hipStream_t stream;
    hipStreamCreateWithFlags(&stream, 0);

    float *hostFloats1;
    hipHostAlloc((void **)&hostFloats1, N * sizeof(float), hipHostMallocPortable);

    hipDeviceptr_t deviceFloats1;
    hipMalloc(&deviceFloats1, N * sizeof(float));

    for(int i = 0; i < 64; i++) {
        hostFloats1[i] = 1000 + i;  // add 1000 so we know we're not just reading eg tid
    }

    hipMemcpyHtoDAsync((hipDeviceptr_t)(((float *)deviceFloats1)), hostFloats1, N * sizeof(float), stream);

    getValue<<<dim3(4,1,1), dim3(128,1,1), 0, stream>>>(((float *)deviceFloats1));

    hipMemcpyDtoHAsync(hostFloats1, deviceFloats1, N * sizeof(float), stream);
    hipStreamSynchronize(stream);

    // and check the values...
    cout << hostFloats1[0] << endl;
    cout << hostFloats1[1] << endl;
    cout << hostFloats1[2] << endl;

    cout << hostFloats1[4] << endl;
    cout << hostFloats1[5] << endl;

    assert(hostFloats1[0] == 1001);
    assert(hostFloats1[1] == 1002);
    assert(hostFloats1[3] == 1004);
    assert(hostFloats1[30] == 1031);
    assert(hostFloats1[31] == 1031);

    hipHostFree(hostFloats1);
    hipFree(deviceFloats1);
    hipStreamDestroy(stream);

    return 0;
}
