// tests cuEventCreate

#include <iostream>
#include <memory>

using namespace std;

#include <hip/hip_runtime.h>

__global__ void longKernel(float *data, int N, float value) {
    for(int i = 0; i < N; i++) {
        data[i] += value;
    }
}

void test1() {
    int N = 102400;

    hipStream_t stream;
    hipStreamCreateWithFlags(&stream, 0);
    cout << "got stream" << endl;

    float *hostFloats;
    hipHostAlloc((void **)&hostFloats, N * sizeof(float), hipHostMallocPortable);

    hipDeviceptr_t deviceFloats;
    hipMalloc(&deviceFloats, N * sizeof(float));

    longKernel<<<dim3(102400 / 32, 1, 1), dim3(32, 1, 1), 0, stream>>>((float *)deviceFloats, N, 3.0f);
    cout << "queued kernel 1" << endl;

    hipEvent_t event;
    hipEventCreateWithFlags(&event, hipEventDisableTiming);
    hipEventRecord(event, stream);
    hipStreamWaitEvent(stream, event, 0);

    longKernel<<<dim3(102400 / 32, 1, 1), dim3(32, 1, 1), 0, stream>>>((float *)deviceFloats, N, 3.0f);
    cout << "queued kernel 2" << endl;

    // cuCtxSynchronize();
    hipStreamSynchronize(stream);
    cout << "finished" << endl;

    hipEventDestroy(event);
    hipHostFree(hostFloats);
    hipFree(deviceFloats);
    hipStreamDestroy(stream    );
}

void dump(float *M, int N) {
    for(int row=0; row < N; row++) {
        cout << " " << M[row];
    }
    cout << endl;
}

void fill(float *M, int N, float val) {
    for(int row=0; row < N; row++) {
        M[row] = val;
    }
}

void test2() {
    // use a long running kernel, queue an async copy back from device
    // => returned values should, in theory, be correct...
    int N = 102400;

    hipStream_t stream;
    hipStreamCreateWithFlags(&stream, 0);

    float hostFloats[N];

    hipDeviceptr_t deviceFloats;
    cout << "call cumemalloc" << endl;
    hipMalloc(&deviceFloats, N * sizeof(float));
    cout << "cumemalloc done" << endl;
    fill(hostFloats, 10, 123);
    dump(hostFloats, 10);

    cout << "calling  hipMemcpyHtoDAsync" << endl;
    hipMemcpyHtoDAsync((hipDeviceptr_t)(((float *)deviceFloats)), hostFloats, N * sizeof(float), stream);
    cout << "hipMemcpyHtoDAsync done" << endl;
    longKernel<<<dim3(102400 / 32, 1, 1), dim3(32, 1, 1), 0, stream>>>((float *)deviceFloats, N, 3.0f);
    cout << "queued kernel" << endl;
    hipMemcpyDtoHAsync(hostFloats, (hipDeviceptr_t)((float *)deviceFloats), N * sizeof(float), stream);
    cout << "queued async copy" << endl;
    hipStreamSynchronize(stream);

    dump(hostFloats, 10);

    hipFree(deviceFloats);
    hipStreamDestroy(stream);
}

int main(int argc, char *argv[]) {
    cout << "test1" << endl;
    test1();
    cout << "test2" << endl;
    test2();

    return 0;
}
