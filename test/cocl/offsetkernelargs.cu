
#include <iostream>
#include <memory>
#include <cassert>

using namespace std;

#include <hip/hip_runtime.h>

__global__ void getValue(float *outdata, float *indata) {
    outdata[0] = indata[0] + 3.0f;
}

int main(int argc, char *argv[]) {
    int N = 1024;

    hipStream_t stream;
    hipStreamCreateWithFlags(&stream, 0);

    float *hostFloats1;
    float *hostFloats2;
    hipHostAlloc((void **)&hostFloats1, N * sizeof(float), hipHostMallocPortable);
    hipHostAlloc((void **)&hostFloats2, N * sizeof(float), hipHostMallocPortable);

    hipDeviceptr_t deviceFloats1;
    hipDeviceptr_t deviceFloats2;
    hipMalloc(&deviceFloats1, N * sizeof(float));
    hipMalloc(&deviceFloats2, N * sizeof(float));

    hostFloats1[128] = 123.456f;

    hipMemcpyHtoDAsync(
        (hipDeviceptr_t)(((float *)deviceFloats1)),
        hostFloats1,
        N * sizeof(float),
        stream
    );
    // cuStreamSynchronize(stream);

    getValue<<<dim3(1,1,1), dim3(32,1,1), 0, stream>>>(((float *)deviceFloats2) + 64, ((float *)deviceFloats1) + 128);

    // now copy back entire buffer
    // hostFloats[64] = 0.0f;
    hipMemcpyDtoHAsync(hostFloats2, deviceFloats2, N * sizeof(float), stream);
    hipStreamSynchronize(stream);

    // and check the values...
    cout << hostFloats2[64] << endl;

    assert(hostFloats2[64] == 126.456f);

    hipHostFree(hostFloats1);
    hipHostFree(hostFloats2);
    hipFree(deviceFloats1);
    hipFree(deviceFloats2);
    hipStreamDestroy(stream);

    return 0;
}
