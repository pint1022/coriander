// tests cuEventCreate

#include <iostream>
#include <memory>

using namespace std;

#include <hip/hip_runtime.h>

__global__ void longKernel(float *data, int N, float value) {
    for(int i = 0; i < N; i++) {
        data[i] += value;
    }
}

int main(int argc, char *argv[]) {
    int N = 102400; // * 1024;

    hipStream_t stream;
    hipStreamCreateWithFlags(&stream, 0);

    float *hostfloats;
    hipHostAlloc((void **)&hostfloats, N * sizeof(float), hipHostMallocPortable);

    hipDeviceptr_t devicefloats;
    hipMalloc(&devicefloats, N * sizeof(float));

    // floats[2] = 4.0f;
    hipMemcpyHtoDAsync(devicefloats, hostfloats, N * sizeof(float), stream);
    longKernel<<<dim3(102400 / 32, 1, 1), dim3(32, 1, 1)>>>((float *)devicefloats, N, 3.0f);
    cout << "queued kernel x" << endl;

    hipCtxSynchronize();
    cout << "finished" << endl;

    hipHostFree(hostfloats);
    hipFree(devicefloats);
    hipStreamDestroy(stream);

    return 0;
}
